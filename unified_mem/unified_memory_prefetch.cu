#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>

__global__ void add(int n, float *x, float *y)
{
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    size_t stride = gridDim.x * blockDim.x;

    for (size_t i = idx; i < n; i += stride)
    {
        y[i] = x[i] + y[i];
    }
}

int main()
{
    int N = 1 << 20;
    float *x, *y;
    int device = -1;

    // allocate unified memory -- accessible from cpu or gpu
    hipMallocManaged(&x, N * sizeof(float));
    hipMallocManaged(&y, N * sizeof(float));

    hipGetDevice(&device);

    hipMemPrefetchAsync(x, N * sizeof(float), device, NULL);
    hipMemPrefetchAsync(y, N * sizeof(float), device, NULL);
    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;
    add<<<numBlocks, blockSize>>>(N, x, y);
    hipMemPrefetchAsync(y, N * sizeof(float), hipCpuDeviceId, NULL);
    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    // Check for errors (all values should be 3.0f)
    float maxError = 0.0f;
    for (int i = 0; i < N; i++)
        maxError = fmax(maxError, fabs(y[i] - 3.0f));
    std::cout << "Max error: " << maxError << std::endl;

    // Free memory
    hipFree(x);
    hipFree(y);

    return 0;
}