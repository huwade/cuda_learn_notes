
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <iostream>

using namespace std;

// error checking macro
#define cudaCheckErrors(msg)                                   \
    do                                                         \
    {                                                          \
        hipError_t __err = hipGetLastError();                \
        if (__err != hipSuccess)                              \
        {                                                      \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                    msg, hipGetErrorString(__err),            \
                    __FILE__, __LINE__);                       \
            fprintf(stderr, "*** FAILED - ABORTING\n");        \
            exit(1);                                           \
        }                                                      \
    } while (0)

struct list_elem
{
    int key;
    list_elem *next;
};

template <typename T>
void alloc_bytes(T &ptr, size_t num_bytes)
{
    hipMallocManaged(&ptr, num_bytes);
}

__host__ __device__ void print_element(list_elem *list)
{
    list_elem *head = list;
    while (head)
    {
        printf("key = %d\n", head->key);
        head = head->next;
    }
}

__global__ void gpu_print_element(list_elem *list)
{
    print_element(list);
}

const int num_elem = 5;

int main()
{

    list_elem *list_base, *list;
    alloc_bytes(list_base, sizeof(list_elem));
    list = list_base;

    for (int i = 0; i < num_elem; i++)
    {
        list->key = i;
        alloc_bytes(list->next, sizeof(list_elem));
        list = list->next;
    }

    print_element(list_base);
    gpu_print_element<<<1, 1>>>(list_base);
    hipDeviceSynchronize();
    cudaCheckErrors("cuda error!");
}
