#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

#include "blocktiling_5.cuh"
#include "cuda_utils.cuh"
#include "naive_0.cuh"
#include "online_1.cuh"
#include "sharedmem_2.cuh"
#include "shfl_3.cuh"
#include "vectorized_4.cuh"

/**
 * Helper function to generate a clamped random number sampled from
 * a normal dist. with mean 0 and std 1.
 * u1, u2 are uniformly distributed random numbers in the range [0.0, 1.0]
 *
 */

float random_normal_clamped(float min, float max)
{
    float u1 = static_cast<float>(rand()) / RAND_MAX;
    float u2 = static_cast<float>(rand()) / RAND_MAX;
    float num = sqrtf(-2.0f * logf(u1)) * cosf(2.0f * M_PI * u2);

    if (num < min)
        return min;

    if (num > max)
        return max;

    return num;
}

int main()
{
    int M = 4096;
    int N = 4096;
    int matsize = M * N;
    int totalsize = matsize * sizeof(float);

    float *mat = (float *)malloc(totalsize);
    float *res = (float *)malloc(totalsize);
    for (int i = 0; i < matsize; i++)
        mat[i] = random_normal_clamped(-10, 10);

    float *matd, *resd;
    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));
    float ms = 0.0f;

    hipEventRecord(start);
    CUDA_CHECK(hipMalloc(&matd, totalsize));
    CUDA_CHECK(hipMalloc(&resd, totalsize));
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);
    printf(">> gpu allocation time %f ms\n", ms);

    hipEventRecord(start);
    CUDA_CHECK(hipMemcpy(matd, mat, totalsize, hipMemcpyHostToDevice));
    hipEventRecord(stop);
    hipEventElapsedTime(&ms, start, stop);
    printf(">> host to device transfer time: %F ms\n", ms);

    run_kernel_4(matd, resd, M, N);
}
