#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

#include "cuda_utils.cuh"

__global__ void softmax_kernel_1(float *__restrict__ matd, float *__restrict__ resd, int M, int N)
{
    int row = blockDim.x * blockIdx.x + threadIdx.x;

    if (row < M)
    {
        float m = -1 * INFINITY;
        float L = 0.0f;

        for (int col = 0; col < N; col++)
        {
            int i = row * N + col;
            float curr = matd[i];
            if (curr > m)
            {
                L = L * expf(m - curr);
                m = curr;
            }
            L += expf(curr - m);
        }
        for (int col = 0; col < N; col++)
        {
            int i = row * N + col;
            resd[i] = expf(matd[i] - m) / L;
        }
    }
}

void run_kernel_1(float *__restrict__ matd, float *__restrict__ resd, int M, int N)
{
    dim3 block_size(1024);
    dim3 grid_size(CEIL_DIV(M, block_size.x));
    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));
    float ms = 0.f;

    CUDA_CHECK(hipEventRecord(start));
    softmax_kernel_1<<<grid_size, block_size>>>(matd, resd, M, N);
    CUDA_CHECK(hipEventRecord(stop));
    CUDA_CHECK(hipEventSynchronize(stop));
    CUDA_CHECK(hipEventElapsedTime(&ms, start, stop));
    printf(">> Kernel execution time: %f ms\n", ms);

    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));
}